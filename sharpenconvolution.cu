#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void evaluate(int x, int* coeffArr, int* outputTerms)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int coefficient = coeffArr[index];
	int result = pow(x, index);
	outputTerms[index] = coefficient*result;
}

#define THREADS_PER_BLOCK 512

void convolve(unsigned char* image, unsigned char* convolvedImage, int height, int width, int channels, float* kernel, int kernelSize) 
{
	int outIndex = 0;
	// Image looping by height x width x color channels
    for(int y = 0; y < height; y++) 
	{
        for(int x = 0; x < width; x++)
		{
			for(int channel = 0; channel < channels; channel++)
			{	
				int radiusx = kernelSize / 2;
				int radiusy = kernelSize / 2;

				float retf = 0.0f;
				float totalWeight = 0.0f;

				for(int iy = -radiusy; iy <= radiusy; iy++)
				{
					int ready = (y + iy + height) % height;
					for(int ix = -radiusx; ix <= radiusx; ix++)
					{
						int readx = (x + ix + width) % width;
						
						float pixelValue = float(image[(ready * width + readx) * channels + channel]) / 255.0f;

						float kernelValue = kernel[(iy + radiusy) * kernelSize + ix + radiusx];

						retf += pixelValue * kernelValue;
						totalWeight += kernelValue;
					}
				}

				retf /= totalWeight;
				convolvedImage[outIndex] = (unsigned char)(fmax(fmin(retf * 256.0f, 255.0f),0.0f));
				outIndex++;
			}
        }
    }
}

int main()
{
	int width, height, channels;
    unsigned char* image;
    if(!(image = stbi_load("./chicago.jpg", &width, &height, &channels, 0))) 
	{
        fprintf(stderr, "Error loading image\n");
        exit(1);
    }

	int N = width*height;
	int k_size = 3;
	float* kernel = new float[k_size*k_size];
	kernel[0] = kernel[2] = kernel[6] = kernel[8] = 0;//0.0023f; // Corners
	kernel[1] = kernel[3] = kernel[5] = kernel[7] = 0;//0.0432f; // Middles
	kernel[4] = 1;//0.8180f; // Center
	
	unsigned char* convolvedImage = stbi_load("./chicago.jpg", &width, &height, &channels, 0);//= (unsigned char*)malloc(N * channels * sizeof(unsigned char*));
    convolve(image, convolvedImage, height, width, channels, kernel, k_size);

	int result; 
    if(result = stbi_write_png("./output.jpg", width, height, channels, convolvedImage, width * channels)) 
	{
		printf("Image saved successfully\n");
		for(int i = 0; i < 10; i++)
		{
			printf("Value check: %d | %d\n", image[i], convolvedImage[i]);
		}
    }
	else 
	{
        printf("Error saving image\n");
    }

    stbi_image_free(image);
	delete kernel;

	// int size = N * sizeof(int);
    // int *coeffArr, *outputTerms;
	// int *d_coeffArr, *d_outputTerms;
	
	// // Allocate space for CPU & GPU arrays
	// hipMalloc((void**) &d_coeffArr, size);
	// hipMalloc((void**) &d_outputTerms, size);
	// coeffArr = (int*)malloc(size);
	// outputTerms = (int*)malloc(size);

	// // Fill coefficient and output arrays
	// for(int i = 0; i < N; i++)
	// {
	// 	coeffArr[i] = i;
	// 	outputTerms[i] = 0;
	// }

	// // Copy coefficient array to GPU
	// hipMemcpy(d_coeffArr, coeffArr, size, hipMemcpyHostToDevice);

	// // Evaluate polynomial using CUDA and copy results back to main memory
	// int x = 1;
	// evaluate<<<(N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(x, d_coeffArr, d_outputTerms);
	// hipMemcpy(outputTerms, d_outputTerms, size, hipMemcpyDeviceToHost);

	// // Sum output
	// int sum = 0;
	// for(int i = 0; i < N; i++)
	// 	sum += outputTerms[i];
	// printf("N = %d\nx = %d\noutputTerms sum = %d\n", N, x, sum);

	// // Clean up
	// free(coeffArr);
	// free(outputTerms);
	// hipFree(d_coeffArr);
	// hipFree(d_outputTerms);
	
	return 0;
}
