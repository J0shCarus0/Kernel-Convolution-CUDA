#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void cuda_convolve(unsigned char* image, unsigned char* convolvedImage, int height, int width, int channels, float* kernel, int kernelSize)
{
	// Thread identifiers
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int c = threadIdx.z;

	if(x < width && y < height && c < channels)
	{
		unsigned char pixelValue = 0.0f;
		for(int rowOffset = -kernelSize / 2; rowOffset <= kernelSize / 2; rowOffset++)
		{
			for(int colOffset = -kernelSize / 2; colOffset <= kernelSize / 2; colOffset++)
			{
				int xCoordImg = x + rowOffset;
				int yCoordImg = y + colOffset;
				int imageIdx = (yCoordImg * width + xCoordImg) * channels + c;

				int kernelIdx = (rowOffset + kernelSize / 2) * kernelSize + (colOffset + kernelSize / 2);

				// Prevent overflow
				if(pixelValue + kernel[kernelIdx] * image[imageIdx] < pixelValue)
				{
					pixelValue = 0xff;
				}
				else
				{
					pixelValue += kernel[kernelIdx] * image[imageIdx];
				}
			}
		}

		int outIdx = (y * width + x) * channels + c;
		convolvedImage[outIdx] = pixelValue;
	}
}

__global__ void evaluate(int x, int* coeffArr, int* outputTerms)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int coefficient = coeffArr[index];
	int result = pow(x, index);
	outputTerms[index] = coefficient*result;
}

#define THREADS_PER_BLOCK 512

void convolve(unsigned char* image, unsigned char* convolvedImage, int height, int width, int channels, float* kernel, int kernelSize) 
{
	int outIndex = 0;
	// Image looping by height x width x color channels
    for(int y = 0; y < height; y++) 
	{
        for(int x = 0; x < width; x++)
		{
			for(int channel = 0; channel < channels; channel++)
			{	
				int radiusx = kernelSize / 2;
				int radiusy = kernelSize / 2;

				float retf = 0.0f;
				float totalWeight = 0.0f;

				for(int iy = -radiusy; iy <= radiusy; iy++)
				{
					int ready = (y + iy + height) % height;
					for(int ix = -radiusx; ix <= radiusx; ix++)
					{
						int readx = (x + ix + width) % width;
						
						float pixelValue = float(image[(ready * width + readx) * channels + channel]) / 255.0f;

						float kernelValue = kernel[(iy + radiusy) * kernelSize + ix + radiusx];

						retf += pixelValue * kernelValue;
						totalWeight += kernelValue;
					}
				}

				retf /= totalWeight;
				convolvedImage[outIndex] = (unsigned char)(fmax(fmin(retf * 256.0f, 255.0f),0.0f));
				outIndex++;
			}
        }
    }
}

int main()
{
	int width, height, channels;
    unsigned char* image;
    if(!(image = stbi_load("./chicago.jpg", &width, &height, &channels, 0))) 
	{
        fprintf(stderr, "Error loading image\n");
        exit(1);
    }

	int N = width*height;
	int k_size = 3;
	float* kernel = new float[k_size*k_size];
	kernel[0] = kernel[2] = kernel[6] = kernel[8] = 0;//0.0023f; // Corners
	kernel[1] = kernel[3] = kernel[5] = kernel[7] = 0;//0.0432f; // Middles
	kernel[4] = 1;//0.8180f; // Center
	
	unsigned char* convolvedImage = stbi_load("./chicago.jpg", &width, &height, &channels, 0);//= (unsigned char*)malloc(N * channels * sizeof(unsigned char*));

	// Allocate GPU memory
	unsigned char* d_image;
	unsigned char* d_convolvedImage;
	float* d_kernel;
	hipMalloc(&d_image, N * channels * sizeof(unsigned char));
	hipMalloc(&d_convolvedImage, N * channels * sizeof(unsigned char));
	hipMalloc(&d_kernel, k_size * k_size * sizeof(float));

	// Copy data into GPU
	hipMemcpy(d_image, image, N * channels * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(d_kernel, kernel, k_size * k_size * sizeof(float), hipMemcpyHostToDevice);

	// Define threads
	dim3 blockSize(3, 3, channels);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

	// Perform operation
	cuda_convolve<<<gridSize, blockSize>>>(d_image, d_convolvedImage, height, width, channels, d_kernel, 3);

	// Wait for operation to complete
	hipDeviceSynchronize();
	
	// Copy data out of GPU
	hipMemcpy(convolvedImage, d_convolvedImage, N * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);

	// Free GPU memory
	hipFree(d_image);
	hipFree(d_convolvedImage);
	hipFree(d_kernel);

	int result; 	
    if(result = stbi_write_png("./cuda_output.jpg", width, height, channels, convolvedImage, width * channels)) 
	{
		printf("Cuda convolved Image saved successfully\n");
		for(int i = 0; i < 10; i++)
		{
			printf("Value check: %d | %d\n", image[i], convolvedImage[i]);
		}
    }
	else 
	{
        printf("Error saving image\n");
    }

	convolve(image, convolvedImage, height, width, channels, kernel, k_size);

    if(result = stbi_write_png("./lin_output.jpg", width, height, channels, convolvedImage, width * channels)) 
	{
		printf("Linear convolved Image saved successfully\n");
		for(int i = 0; i < 10; i++)
		{
			printf("Value check: %d | %d\n", image[i], convolvedImage[i]);
		}
    }
	else 
	{
        printf("Error saving image\n");
    }

    stbi_image_free(image);
	delete kernel;

	// int size = N * sizeof(int);
    // int *coeffArr, *outputTerms;
	// int *d_coeffArr, *d_outputTerms;
	
	// // Allocate space for CPU & GPU arrays
	// hipMalloc((void**) &d_coeffArr, size);
	// hipMalloc((void**) &d_outputTerms, size);
	// coeffArr = (int*)malloc(size);
	// outputTerms = (int*)malloc(size);

	// // Fill coefficient and output arrays
	// for(int i = 0; i < N; i++)
	// {
	// 	coeffArr[i] = i;
	// 	outputTerms[i] = 0;
	// }

	// // Copy coefficient array to GPU
	// hipMemcpy(d_coeffArr, coeffArr, size, hipMemcpyHostToDevice);

	// // Evaluate polynomial using CUDA and copy results back to main memory
	// int x = 1;
	// evaluate<<<(N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(x, d_coeffArr, d_outputTerms);
	// hipMemcpy(outputTerms, d_outputTerms, size, hipMemcpyDeviceToHost);

	// // Sum output
	// int sum = 0;
	// for(int i = 0; i < N; i++)
	// 	sum += outputTerms[i];
	// printf("N = %d\nx = %d\noutputTerms sum = %d\n", N, x, sum);

	// // Clean up
	// free(coeffArr);
	// free(outputTerms);
	// hipFree(d_coeffArr);
	// hipFree(d_outputTerms);
	
	return 0;
}
